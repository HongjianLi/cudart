
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>

int main(int argc, char *argv[])
{
	// Initialize constants.
	int block_size = 32;
	int wA = 2 * block_size * 5;
	int hA = 4 * block_size * 5;
	int wB = 2 * block_size * 5;
	int hB = 4 * block_size * 5;
	int wC = 2 * block_size * 5;
	int hC = 4 * block_size * 5;
	unsigned int size_A = wA * hA;
	unsigned int size_B = wB * hB;
	unsigned int size_C = wC * hC;
	unsigned int mem_size_A = sizeof(float) * size_A;
	unsigned int mem_size_B = sizeof(float) * size_B;
	unsigned int mem_size_C = sizeof(float) * size_C;

	// Allocates matrices a, b and c in host memory.
	float *h_A = (float *)malloc(mem_size_A);
	float *h_B = (float *)malloc(mem_size_B);
	float *h_C = (float *)malloc(mem_size_C);

	// Initialize matrices a and b.
	srand(2006);
	for (int i = 0; i < size_A; ++i)
	{
		h_A[i] = rand() / (float)RAND_MAX;
	}
	for (int i = 0; i < size_B; ++i)
	{
		h_B[i] = rand() / (float)RAND_MAX;
	}

	// Allocate matrices a, b and c in device memory.
	float *d_A, *d_B, *d_C;
	hipMalloc((void **)&d_A, mem_size_A);
	hipMalloc((void **)&d_B, mem_size_B);
	hipMalloc((void **)&d_C, mem_size_C);

	// Copy matrices a and b from host memory to device memory.
	hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

	// Determine the number of threads per block and the number of blocks per grid.
	dim3 numThreadsPerBlock(block_size, block_size);
	dim3 numBlocksPerGrid(wC / numThreadsPerBlock.x, hC / numThreadsPerBlock.y);

	// Initialize a cublas handle.
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// CUBLAS is column primary.
	const float alpha = 1.0f;
	const float beta  = 0.0f;
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, wB, hA, wA, &alpha, d_B, wB, d_A, wA, &beta, d_C, wA);

	// Measure the performance of cublasSgemm over a number of iterations.
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	int numIterations = 30;
	for (int i = 0; i < numIterations; ++i)
	{
		hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, wB, hA, wA, &alpha, d_B, wB, d_A, wA, &beta, d_C, wA);
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsed;
	hipEventElapsedTime(&elapsed, start, stop);

	// Compute and print the GLOPS/s performance metric.
	printf("%.2f GFLOP/s\n", (2.0f * wA * hA * wB * numIterations * 1e-9f) / (elapsed / 1000.0f));

	// Copy matrix c from device memory to host memory.
	hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

	// Compute reference solution.
	float *ref = (float *)malloc(mem_size_C);
	for (int i = 0; i < hA; ++i)
	{
		for (int j = 0; j < wB; ++j)
		{
			float sum = 0;
			for (int k = 0; k < wA; ++k)
			{
				sum += h_A[i * wA + k] * h_B[k * wB + j];
			}
			ref[i * wB + j] = sum;
		}
	}

	// Validate the result.
	for (int i = 0; i < size_C; ++i)
	{
		float actual = h_C[i];
		float expected = ref[i];
		if (fabs(actual - expected) / fabs(actual) / wA > 1e-7)
		{
			printf("h_C[%d] = %f, expected = %f\n", i, actual, expected);
			break;
		}
	}

	// Cleanup.
	free(ref);
	hipblasDestroy(handle);
	hipFree(d_C);
	hipFree(d_B);
	hipFree(d_A);
	hipDeviceReset();
	free(h_C);
	free(h_B);
	free(h_A);
}
