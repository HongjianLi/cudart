
#include <hip/hip_runtime.h>
#include <stdio.h>

template <int BLOCK_SIZE> __global__ void matrixMulCUDA(float *C, float *A, float *B, int wA, int wB)
{
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int aBegin = wA * BLOCK_SIZE * by;
	int aEnd   = aBegin + wA - 1;
	int aStep  = BLOCK_SIZE;
	int bBegin = BLOCK_SIZE * bx;
	int bStep  = BLOCK_SIZE * wB;
	float Csub = 0;
	for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep)
	{
		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
		As[ty][tx] = A[a + wA * ty + tx];
		Bs[ty][tx] = B[b + wB * ty + tx];
		__syncthreads();
#pragma unroll
		for (int k = 0; k < BLOCK_SIZE; ++k)
		{
			Csub += As[ty][k] * Bs[k][tx];
		}
		__syncthreads();
	}
	int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
	C[c + wB * ty + tx] = Csub;
}

void constantInit(float *data, int size, float val)
{
	for (int i = 0; i < size; ++i)
	{
		data[i] = val;
	}
}

int main(int argc, char **argv)
{
	const int block_size = 32;
	dim3 dimsA(5 * 2 * block_size, 5 * 2 * block_size, 1);
	dim3 dimsB(5 * 4 * block_size, 5 * 2 * block_size, 1);
	unsigned int size_A = dimsA.x * dimsA.y;
	unsigned int mem_size_A = sizeof(float) * size_A;
	float *h_A = (float *)malloc(mem_size_A);
	unsigned int size_B = dimsB.x * dimsB.y;
	unsigned int mem_size_B = sizeof(float) * size_B;
	float *h_B = (float *)malloc(mem_size_B);
	for (int i = 0; i < size_A; ++i)
	{
		h_A[i] = 1.0f;
	}
	const float valB = 0.01f;
	for (int i = 0; i < size_B; ++i)
	{
		h_B[i] = valB;
	}
	dim3 dimsC(dimsB.x, dimsA.y, 1);
	unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
	float *h_C = (float *)malloc(mem_size_C);
	float *d_A, *d_B, *d_C;
	hipMalloc((void **)&d_A, mem_size_A);
	hipMalloc((void **)&d_B, mem_size_B);
	hipMalloc((void **)&d_C, mem_size_C);
	hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);
	dim3 threads(block_size, block_size);
	dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);
	matrixMulCUDA<block_size><<<grid, threads>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
	hipDeviceSynchronize();

	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	int nIter = 300;
	for (int j = 0; j < nIter; j++)
	{
		matrixMulCUDA<block_size><<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float msecTotal;
	hipEventElapsedTime(&msecTotal, start, stop);

	float msecPerMatrixMul = msecTotal / nIter;
	double flopsPerMatrixMul = 2.0 * (double)dimsA.x * (double)dimsA.y * (double)dimsB.x;
	double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
	printf("Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops, WorkgroupSize= %u threads/block\n", gigaFlops, msecPerMatrixMul, flopsPerMatrixMul, threads.x * threads.y);
	hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

	double eps = 1.e-6 ;
	for (int i = 0; i < (int)(dimsC.x * dimsC.y); i++)
	{
		double abs_err = fabs(h_C[i] - (dimsA.x * valB));
		double dot_length = dimsA.x;
		double abs_val = fabs(h_C[i]);
		double rel_err = abs_err/abs_val/dot_length ;
		if (rel_err > eps)
		{
			printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n", i, h_C[i], dimsA.x*valB, eps);
		}
	}
	free(h_A);
	free(h_B);
	free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipDeviceReset();
}
