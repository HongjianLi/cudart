#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void hyperQ(clock_t num_clocks)
{
	for (const clock_t threshold = clock() + num_clocks; clock() < threshold;);
}

int main(int argc, char *argv[])
{
	const int num_milliseconds = 10;
	const int num_kernels = 2;
	const int num_streams = 32;
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	clock_t num_clocks = deviceProp.clockRate * num_milliseconds;
	hipStream_t *streams = (hipStream_t*)malloc(sizeof(hipStream_t) * num_streams);
	for (int s = 0; s < num_streams; ++s)
	{
		hipStreamCreate(&streams[s]);
	}
	hipEvent_t beg, end;
	hipEventCreate(&beg);
	hipEventCreate(&end);
	hipEventRecord(beg, 0);
	for (int s = 0; s < num_streams; ++s)
	{
		for (int k = 0; k < num_kernels; ++k)
		{
			hyperQ<<<1, 1, 0, streams[s]>>>(num_clocks);
		}
	}
	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	float elapsed;
	hipEventElapsedTime(&elapsed, beg, end);
	hipEventDestroy(end);
	hipEventDestroy(beg);
	for (int s = 0; s < num_streams; ++s)
	{
		hipStreamDestroy(streams[s]);
	}
	free(streams);
	hipDeviceReset();
	printf("%d streams, each %d kernels, each %d ms\n", num_streams, num_kernels, num_milliseconds);
	printf("       SM <= 1.3:%4d ms\n", num_milliseconds * num_kernels * num_streams);
	printf("2.0 <= SM <= 3.0:%4d ms\n", num_milliseconds * (1 + (num_kernels - 1) * num_streams));
	printf("3.5 <= SM       :%4d ms\n", num_milliseconds * num_kernels);
	printf("       SM == %d.%d:%4d ms\n", deviceProp.major, deviceProp.minor, (int)elapsed);
}
