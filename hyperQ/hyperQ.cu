#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

__global__ void hyperQ(clock_t num_clocks)
{
	for (const clock_t threshold = clock() + num_clocks; clock() < threshold;);
}

int main(int argc, char *argv[])
{
	const int num_milliseconds = 10;
	const int num_kernels = 2;
	const int num_streams = 32;
	hipStream_t *streams;
	hipEvent_t beg, end;
	clock_t num_clocks;
	float elapsed;
	int s, k;
    int cuda_device = 0;
    hipDeviceProp_t deviceProp;
    hipGetDevice(&cuda_device);
    hipGetDeviceProperties(&deviceProp, cuda_device);
	num_clocks = deviceProp.clockRate * num_milliseconds;
	streams = (hipStream_t*)malloc(sizeof(hipStream_t) * num_streams);
	for (s = 0; s < num_streams; ++s)
	{
		hipStreamCreate(&streams[s]);
	}
	hipEventCreate(&beg);
	hipEventCreate(&end);
	hipEventRecord(beg, 0);
	for (s = 0; s < num_streams; ++s)
	{
		for (k = 0; k < num_kernels; ++k)
		{
			hyperQ<<<1, 1, 0, streams[s]>>>(num_clocks);
		}
	}
	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsed, beg, end);
	hipEventDestroy(end);
	hipEventDestroy(beg);
	for (s = 0; s < num_streams; ++s)
	{
		hipStreamDestroy(streams[s]);
	}
	free(streams);
    hipDeviceReset();
	printf("%d streams, each %d kernels, each %d ms\n", num_streams, num_kernels, num_milliseconds);
	printf("       SM <= 1.3:%4d ms\n", num_milliseconds * num_kernels * num_streams);
	printf("2.0 <= SM <= 3.0:%4d ms\n", num_milliseconds * (1 + (num_kernels - 1) * num_streams));
	printf("3.5 <= SM       :%4d ms\n", num_milliseconds * num_kernels);
	printf("       SM == %d.%d:%4d ms\n", deviceProp.major, deviceProp.minor, (int)elapsed);
}
