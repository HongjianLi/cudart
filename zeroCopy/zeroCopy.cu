
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void zeroCopy(const float* a, const float* b, float* c)
{
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	c[gid] = a[gid] + b[gid];
}

int main(int argc, char *argv[])
{
	const unsigned int lws = 256;
	const unsigned int gws = 1024 * lws;

	// Allocate pinned vectors a, b and c in host memory with the cudaHostAllocMapped flag.
	float* h_a;
	float* h_b;
	float* h_c;
	hipHostAlloc((void**)&h_a, sizeof(float) * gws, hipHostMallocMapped);
	hipHostAlloc((void**)&h_b, sizeof(float) * gws, hipHostMallocMapped);
	hipHostAlloc((void**)&h_c, sizeof(float) * gws, hipHostMallocMapped);

	// Initialize vectors a and b.
	for (int i = 0; i < gws; ++i)
	{
		h_a[i] = rand() / (float)RAND_MAX;
		h_b[i] = rand() / (float)RAND_MAX;
	}

	// Get the mapped device points.
	float* d_a;
	float* d_b;
	float* d_c;
	hipHostGetDevicePointer(&d_a, h_a, 0);
	hipHostGetDevicePointer(&d_b, h_b, 0);
	hipHostGetDevicePointer(&d_c, h_c, 0);

	// Invoke the kernel.
	zeroCopy<<<gws / lws, lws>>>(d_a, d_b, d_c);
	hipDeviceSynchronize();

	// Validate result.
	for (int i = 0; i < gws; ++i)
	{
		const float ref = h_a[i] + h_b[i];
		if (fabs(h_c[i] - ref) > 1e-7)
		{
			printf("i = %d, ref = %f, h_c[i] = %f\n", i, ref, h_c[i]);
			break;
		}
	}

	// Cleanup.
	hipHostFree(h_c);
	hipHostFree(h_b);
	hipHostFree(h_a);
}
