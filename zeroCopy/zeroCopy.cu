
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void zeroCopy(const float* a, const float* b, float* c)
{
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	c[gid] = a[gid] + b[gid];
}

int main(int argc, char *argv[])
{
	const unsigned int lws = 256;
	const unsigned int gws = 1024 * lws;
	float* h_a;
	float* h_b;
	float* h_c;
	hipHostAlloc((void**)&h_a, sizeof(float) * gws, hipHostMallocMapped);
	hipHostAlloc((void**)&h_b, sizeof(float) * gws, hipHostMallocMapped);
	hipHostAlloc((void**)&h_c, sizeof(float) * gws, hipHostMallocMapped);
	for (int i = 0; i < gws; ++i)
	{
		h_a[i] = rand() / (float)RAND_MAX;
		h_b[i] = rand() / (float)RAND_MAX;
	}
	float* d_a;
	float* d_b;
	float* d_c;
	hipHostGetDevicePointer(&d_a, h_a, 0);
	hipHostGetDevicePointer(&d_b, h_b, 0);
	hipHostGetDevicePointer(&d_c, h_c, 0);
	zeroCopy<<<gws / lws, lws>>>(d_a, d_b, d_c);
	hipDeviceSynchronize();
	bool passed = true;
	for (int i = 0; i < gws; ++i)
	{
		const float ref = h_a[i] + h_b[i];
		if (fabs(h_c[i] - ref) > 1e-7)
		{
			printf("i = %d, ref = %f, h_c[i] = %f\n", i, ref, h_c[i]);
			passed = false;
			break;
		}
	}
	printf("vectorAdd %s\n\n", passed ? "passed" : "failed");
	hipHostFree(h_a);
	hipHostFree(h_b);
	hipHostFree(h_c);
}
