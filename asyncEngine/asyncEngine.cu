
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAdd(const float *a, const float *b, float *c, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < numElements)
	{
		c[i] = a[i] + b[i];
	}
}

int main(int argc, char *argv[])
{
	int numElements = 5e+4;

	// Allocate vectors a, b and c in host memory.
	size_t numBytes = sizeof(float) * numElements;
	float *h_a;
	float *h_b;
	float *h_c;
	hipHostMalloc((void **)&h_a, numBytes, hipHostMallocDefault);
	hipHostMalloc((void **)&h_b, numBytes, hipHostMallocDefault);
	hipHostMalloc((void **)&h_c, numBytes, hipHostMallocDefault);

	// Initialize vectors a and b.
	for (int i = 0; i < numElements; ++i)
	{
		h_a[i] = rand() / (float)RAND_MAX;
		h_b[i] = rand() / (float)RAND_MAX;
	}

	// Initialize a number of CUDA streams.
	int numStreams = 8;
    hipStream_t *streams = (hipStream_t *)malloc(sizeof(hipStream_t) * numStreams);
    for (int i = 0; i < numStreams; ++i)
    {
        hipStreamCreate(&streams[i]);
    }

	// Compute the average number of elements per device and the number of spare elements.
	int avgElementsPerStream = numElements / numStreams;
	int sprElements = numElements - avgElementsPerStream * numStreams;

	float **d_a = (float **)malloc(sizeof(float *) * numStreams);
	float **d_b = (float **)malloc(sizeof(float *) * numStreams);
	float **d_c = (float **)malloc(sizeof(float *) * numStreams);

	for (int i = 0, offset = 0; i < numStreams; ++i)
	{
		// Determine the number of elements to be processed by the current device.
		int numElementsCurrentStream = avgElementsPerStream + (i < sprElements);

		// Allocate vectors a, b and c in device memory.
		size_t numBytesCurrentStream = sizeof(int) * numElementsCurrentStream;
		hipMalloc((void **)&d_a[i], numBytesCurrentStream);
		hipMalloc((void **)&d_b[i], numBytesCurrentStream);
		hipMalloc((void **)&d_c[i], numBytesCurrentStream);

		// Copy vectors a and b from host memory to device memory asynchronously.
		hipMemcpyAsync(d_a[i], h_a + offset, numBytesCurrentStream, hipMemcpyHostToDevice, streams[i]);
		hipMemcpyAsync(d_b[i], h_b + offset, numBytesCurrentStream, hipMemcpyHostToDevice, streams[i]);

		// Determine the number of threads per block and the number of blocks per grid.
		unsigned int numThreadsPerBlock = 256;
		unsigned int numBlocksPerGrid = (numElementsCurrentStream + numThreadsPerBlock - 1) / numThreadsPerBlock;

		// Invoke the kernel on device asynchronously.
		vectorAdd<<<numBlocksPerGrid, numThreadsPerBlock, 0, streams[i]>>>(d_a[i], d_b[i], d_c[i], numElementsCurrentStream);

		// Copy vector c from device memory to host memory asynchronously.
		hipMemcpyAsync(h_c + offset, d_c[i], numBytesCurrentStream, hipMemcpyDeviceToHost, streams[i]);

		// Increase offset to point to the next portion of data.
		offset += numElementsCurrentStream;
	}

	// Wait for the device to finish.
	hipDeviceSynchronize();

	// Validate the result.
	for (int i = 0; i < numElements; ++i)
	{
		float actual = h_c[i];
		float expected = h_a[i] + h_b[i];
		if (fabs(actual - expected) > 1e-7)
		{
			printf("h_c[%d] = %f, expected = %f\n", i, actual, expected);
			break;
		}
	}

	// Cleanup.
	for (int i = 0; i < numStreams; ++i)
	{
		hipFree(d_c[i]);
		hipFree(d_b[i]);
		hipFree(d_a[i]);
	}
	free(d_c);
	free(d_b);
	free(d_a);
    for (int i = 0; i < numStreams; ++i)
    {
        hipStreamDestroy(streams[i]);
    }
    free(streams);
	hipHostFree(h_c);
	hipHostFree(h_b);
	hipHostFree(h_a);
	hipDeviceReset();
}
